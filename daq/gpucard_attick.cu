#include "hip/hip_runtime.h"
/***********************************
***********************************
CUDA PART
***********************************
**********************************/





/* Basic parameters of ADC */
#define SAMPLING_RATE 2.7e9
#define DELTA_T (1.0/SAMPLING_RATE)

/* SIZE OF FFT, BUFFERS 
we aim at 1ms = 2.7MSamples, 
2**21 is 0.8 ms 
2**23 is 0.99ms
*/
#define BUFFER_SIZE (268435456)
#define NUM_FFT 1
#define FFT_SIZE (BUFFER_SIZE/NUM_FFT)
#define TRANSFORM_SIZE (FFT_SIZE/2+1)

/* Size of a chunk we process in one go
At 2.7 MS, this 99.4 ms*/

#define DELTA_NU (1./(DELTA_T*FFT_SIZE))

/* numin and numax in freq in MHZ */
#define NUMIN 600.0
#define NUMAX 1300.0
/* We average over this many FFT bins */
#define NUAVG (16384/NUM_FFT)
/* and we get this many bins */


#define FLOATIZE_X 8
#define THREADS_DEFAULT 1024

void print_settings();
inline int num_nubins() { return (floor((NUMAX-NUMIN)*1e6/DELTA_NU/NUAVG)+1); }



#include <memory.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <assert.h>

#include "gpucard.h"


#include <stdio.h>
#include "math.h"

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define CHK( err ) (HandleError( err, __FILE__, __LINE__ ))


uint8_t* alloc_sample_buffer() {
  uint8_t *p;
  /* size of uint8_t is one, explicityly */
  CHK(hipHostAlloc(&p, BUFFER_SIZE, hipHostMallocDefault));
 return p;
}

float* alloc_power() {
  float *p;
  /* size of uint8_t is one, explicityly */
  CHK(hipHostAlloc(&p, num_nubins()*sizeof(float), hipHostMallocDefault));
  return p;
}


void print_timing (hipEvent_t* start, hipEvent_t* stop, const char* what) {
  float gpu_time;
  CHK(hipEventElapsedTime(&gpu_time, *start, *stop));
  printf ("Timing %s : %fms \n",what, gpu_time);
}


/**
 * CUDA Kernel byte->float
 *
 */
__global__ void floatize(uint8_t* sample,hipfftReal* fsample)  {
    int i = FLOATIZE_X*(blockDim.x * blockIdx.x + threadIdx.x);
    for (int j=0; j<FLOATIZE_X; j++) fsample[i+j]=float(sample[i+j]-128);
}

/**
 * CUDA reduction sum
 * we will take bsize complex numbers starting at ffts[istart+bsize*blocknumber]
 * and their copies in NCHUNS, and add the squares
 **/


__global__ void ps_reduce(hipfftComplex *ffts, float* output_ps, size_t istart) {
  int tid=threadIdx.x;
  int bl=blockIdx.x;
  int nth=blockDim.x;
  __shared__ float work[1024];
//  assert (tid<NUAVG);
  //global pos
  size_t pos=istart+bl*NUAVG+tid;
  //chunk pos
  size_t cpos=tid;
  work[tid]=0;
  size_t chunk=0;
  while (chunk<NUM_FFT) {
//    assert (pos<NUM_FFT*TRANSFORM_SIZE);
    work[tid]+=ffts[pos].x*ffts[pos].x+ffts[pos].y*ffts[pos].y;
    if (cpos+nth<NUAVG) {
      cpos+=nth;
      pos+=nth;
    } else {
      chunk++;
      pos=chunk*TRANSFORM_SIZE+istart+bl*NUAVG+tid;
      cpos=tid;
    }
  }

  // now do the three reduce.
  int csum=nth/2;
  while (csum>0) {
    __syncthreads();
    if (tid<csum) {
      work[tid]+=work[tid+csum];
    }
    csum/=2;
  }
  if (tid==0) output_ps[bl]=work[0];
}
  


void cuda_test(uint8_t *buf, float* freq, float*power) {

  // cuda buffer and float buffer
  uint8_t *cbuf;
  CHK(hipMalloc(&cbuf,BUFFER_SIZE));
  hipfftReal *cfbuf;
  CHK(hipMalloc(&cfbuf,BUFFER_SIZE*sizeof(hipfftReal)));
  hipfftComplex *ffts;
  CHK(hipMalloc(&ffts,TRANSFORM_SIZE*NUM_FFT*sizeof(hipfftComplex)));
  int istart=int(NUMIN*1e6/DELTA_NU)-NUAVG/2;
  for (size_t i=0;i<num_nubins();i++) freq[i]=(istart+i*NUAVG)*DELTA_NU/1e6;
  // device power
  float *cpower;
  CHK(hipMalloc(&cpower,num_nubins()*sizeof(float)));
  
  
  hipfftHandle plan;
  //int oembed=TRANSFORM_SIZE*NUM_FFT+1;
  int fftsize=FFT_SIZE;
  int status=hipfftPlanMany(&plan, 1, &fftsize, NULL, 0, 0, 
        NULL, TRANSFORM_SIZE,1, HIPFFT_R2C, NUM_FFT);
  if (status!=HIPFFT_SUCCESS) {
       printf ("Plan failed:");
       if (status==HIPFFT_ALLOC_FAILED) printf("HIPFFT_ALLOC_FAILED");
       if (status==HIPFFT_INVALID_VALUE) printf ("HIPFFT_INVALID_VALUE");
       if (status==HIPFFT_INTERNAL_ERROR) printf ("HIPFFT_INTERNAL_ERROR");
       if (status==HIPFFT_SETUP_FAILED) printf ("HIPFFT_SETUP_FAILED");
       if (status==HIPFFT_INVALID_SIZE) printf ("HIPFFT_INVALID_SIZE");
       printf("\n");
       exit(1);
  }

  hipEvent_t tstart, tcpy,tfloatize,tfft,treduce,tcopyback;
  CHK(hipEventCreate(&tstart));
  CHK(hipEventCreate(&tcpy));
  CHK(hipEventCreate(&tfloatize));
  CHK(hipEventCreate(&tfft));
  CHK(hipEventCreate(&treduce));
  CHK(hipEventCreate(&tcopyback));

  hipEventRecord(tstart, 0);
  // copy to device
  CHK(hipMemcpy(cbuf,buf, BUFFER_SIZE, hipMemcpyHostToDevice));

  hipEventRecord(tcpy, 0);
  
  // floatize
  int threadsPerBlock = THREADS_DEFAULT;
  int blocksPerGrid = BUFFER_SIZE / threadsPerBlock/FLOATIZE_X;
  floatize<<<blocksPerGrid, threadsPerBlock>>>(cbuf,cfbuf);
  hipEventRecord(tfloatize, 0);
  CHK(hipGetLastError());
  
  status=hipfftExecR2C(plan, cfbuf, ffts);
  hipEventRecord(tfft, 0);
  if (status!=HIPFFT_SUCCESS) {
     printf("CUFFT FAILED\n");
     exit(1);
  }    


  // now launch the final kernel
  while (threadsPerBlock>NUAVG) threadsPerBlock/=2;
  // ps_reduce<<<num_nubins(),threadsPerBlock>>>(ffts,cpower, istart);
  ps_reduce<<<num_nubins(), threadsPerBlock>>>(ffts,cpower, istart);
  hipEventRecord(treduce, 0);
  // copy results over
  CHK(hipMemcpy(power,cpower, num_nubins()*sizeof(float), hipMemcpyDeviceToHost));
  hipEventRecord(tcopyback, 0);

  hipDeviceSynchronize();
  print_timing(&tstart,&tcpy,"MEM CPY");
  print_timing(&tcpy,&tfloatize,"FLOATIZE");
  print_timing(&tfloatize,&tfft,"FFT");
  print_timing(&tfft,&treduce,"REDUCE");
  print_timing(&tfft,&treduce,"COPYBACK");

#ifdef DEBUGREDUCE
  hipfftComplex *hffts;
  CHK(hipHostMalloc(&hffts,TRANSFORM_SIZE*NUM_FFT*sizeof(hipfftComplex)));
  CHK(hipMemcpy(hffts,ffts, TRANSFORM_SIZE*NUM_FFT*sizeof(hipfftComplex), hipMemcpyDeviceToHost));
  //now check first and last elements of the transform which should be real
  for (size_t i=0;i<NUM_FFT;i++) {
    hipfftComplex f=hffts[i*TRANSFORM_SIZE];
    hipfftComplex s=hffts[i*TRANSFORM_SIZE+1];
    hipfftComplex ml=hffts[(i+1)*TRANSFORM_SIZE-2];
    hipfftComplex l=hffts[(i+1)*TRANSFORM_SIZE-1];
    printf ("%i first %f %f , second %f %f, lastbyone %f %f, last %f %f \n",
	    (int)i, f.x,f.y,s.x,s.y,ml.x,ml.y, l.x,l.y);
  }
  //now do the powers
  for (size_t i=0;i<num_nubins();i++) {
    float pow=0;
    for (size_t j=0;j<NUM_FFT;j++) {
      for (size_t k=0;k<NUAVG;k++) {
        int pos=j*TRANSFORM_SIZE+istart+i*NUAVG+k;
	if (!(pos<TRANSFORM_SIZE*NUM_FFT)) {
	  printf ("SHIT %i %i %i %i\n",NUM_FFT, TRANSFORM_SIZE, NUAVG, num_nubins());
	  exit(1);
	}
	pow+=hffts[pos].x*hffts[pos].x+hffts[pos].y*hffts[pos].y;
      }
    }
    printf ("power %i %fMHz %f %f\n", (int)i,freq[i], pow, power[i]);
  }

#endif
}


void ztest() {
  uint8_t *cbuf;
  CHK(hipMalloc(&cbuf,BUFFER_SIZE));
  // floatize
  int threadsPerBlock = 1024;
  int blocksPerGrid = 32768;
  int Nth=
  printf ("%i %i",threadsPerBlock, blocksPerGrid);
  //  floatize<<<blocksPerGrid, threadsPerBlock>>>(cbuf);
  CHK(hipGetLastError());

}
