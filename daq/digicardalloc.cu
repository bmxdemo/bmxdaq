#include <hip/hip_runtime.h>
#include "digicard.h"

//#ifdef USE_DIGICARD_ALLOC 
//#define ALLOC(data, size) \
//    (data) = (int16*) pvAllocMemPageAligned ((uint64) (size);	
//#else
//#define ALLOC(data, size) \
//     if(hipHostMalloc(&data, size)!=hipSuccess){\
//	 printf("Error allocating memory for digitizer buffer.\n");\
//         exit(1);\
//     }
//#endif

void digiCardAlloc(int16* & data, int32 size){
    //ALLOC(data, size);
    if(hipHostMalloc(&data, size)!= hipSuccess){
	printf("Digitizer memory allocation failed\n");
	exit(1);
    }
}
     
