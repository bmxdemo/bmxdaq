#include <hip/hip_runtime.h>
#include "digicardalloc.h"

//#ifdef USE_DIGICARD_ALLOC 
//#define ALLOC(data, size) \
//    (data) = (int16*) pvAllocMemPageAligned ((uint64) (size);	
//#else
//#define ALLOC(data, size) \
//     if(hipHostMalloc(&data, size)!=hipSuccess){\
//	 printf("Error allocating memory for digitizer buffer.\n");\
//         exit(1);\
//     }
//#endif

void digiCardAlloc(int16* & data, size_t size){
    //ALLOC(data, size);
    if(hipHostMalloc(&data, size)!= hipSuccess){
	printf("Digitizer memory allocation failed\n");
	printf ("Requested %li bytes = %i Gb.\n",size, size/1024/1024/1024);
	exit(1);
    }
}
     
void digiCardFree(int16* & data){
    if(hipHostFree(data)!= hipSuccess){
	printf("Freeing of digitizer memory failed\n");
	exit(1);
    }
}
