#include "hip/hip_runtime.h"
/***********************************
***********************************
CUDA PART
***********************************
**********************************/

#include "gpucard.h"
#include "terminal.h"

#include <memory.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <math.h>

#define FLOATIZE_X 2


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
      printf( "CUDA fail: %s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit(1);
    }
}
#define CHK( err ) (HandleError( err, __FILE__, __LINE__ ))

void gpuCardInit (GPUCARD *gc, SETTINGS *set) {
  printf ("\n\nInitializing GPU\n");
  printf ("====================\n");
  printf ("Allocating GPU buffers\n");
  int Nb=set->cuda_streams;
  gc->cbuf=(void**)malloc(Nb*sizeof(void*));
  gc->cfbuf=(void**)malloc(Nb*sizeof(void*));
  gc->cfft=(void**)malloc(Nb*sizeof(void*));
  gc->coutps=(void**)malloc(Nb*sizeof(void*));
  int nchan=gc->nchan=1+(set->channel_mask==3);
  if ((nchan==2) and (FLOATIZE_X%2==1)) {
    printf ("Need FLOATIZE_X even for two channels\n");
    exit(1);
  }
  gc->fftsize=set->fft_size;
  uint32_t bufsize=gc->bufsize=set->fft_size*nchan;
  uint32_t transform_size=(set->fft_size/2+1)*nchan;
  float nunyq=set->sample_rate/2;
  float dnu=nunyq/(set->fft_size/2+1);
  gc->tot_pssize=0;
  gc->ncuts=set->n_cuts;
  for (int i=0; i<gc->ncuts; i++) {
    printf ("Cutout %i:\n",i);
    gc->fftavg[i]=set->fft_avg[i];
    // first sort  reflections etc.
    //
    float numin, numax;
    numin=set->nu_min[i];
    numax=set->nu_max[i];
    while (fabs(numin)>nunyq) numin-=set->sample_rate;
    while (fabs(numax)>nunyq) numax-=set->sample_rate;
    numin=abs(numin);
    numax=abs(numax);
    if (numax<numin) { float t=numin; numin=numax; numax=t; }
    printf ("   Frequencies %f - %f Mhz appear as %f - %f \n",set->nu_min[i]/1e6, set->nu_max[i]/1e6,
	    numin/1e6, numax/1e6);
    int imin=int(numin/dnu);
    if (imin==0) imin=1;
    int imax=int(numax/dnu)+1;
    gc->pssize1[i]=(imax-imin)/set->fft_avg[i];
    gc->ndxofs[i]=imin;
    if ((imax-imin)%set->fft_avg[i]>0) gc->pssize1[i]+=1;
    imax=imin+gc->pssize1[i]*set->fft_avg[i];
    numin=imin*dnu;
    numax=imax*dnu;
    set->nu_min[i]=numin;
    set->nu_max[i]=numax;
    set->pssize[i]=gc->pssize1[i];
    if (nchan==2)
      gc->pssize[i]=gc->pssize1[i]*4; // for other and two crosses
    else
      gc->pssize[i]=gc->pssize1[i]; // just one power spectrum
    gc->tot_pssize+=gc->pssize[i];
    printf ("   Actual freq range: %f - %f MHz (edges!)\n",numin/1e6, numax/1e6);
    printf ("   # PS offset, #PS bins: %i %i\n",gc->ndxofs[i],gc->pssize1[i]);
  }
  
  for (int i=0;i<Nb;i++) {
    uint8_t** cbuf=(uint8_t**)&(gc->cbuf[i]);
    CHK(hipMalloc(cbuf,bufsize));
    hipfftReal** cfbuf=(hipfftReal**)&(gc->cfbuf[i]);
    CHK(hipMalloc(cfbuf, bufsize*sizeof(hipfftReal)));
    hipfftComplex** ffts=(hipfftComplex**)&(gc->cfft[i]);
    CHK(hipMalloc(ffts,transform_size*sizeof(hipfftComplex)));
    float** coutps=(float**)&(gc->coutps[i]);
    CHK(hipMalloc(coutps,gc->tot_pssize*sizeof(float)));
  }
  CHK(hipHostAlloc(&gc->outps, gc->tot_pssize*sizeof(float), hipHostMallocDefault));

  printf ("Setting up CUFFT\n");
  int status=hipfftPlanMany(&gc->plan, 1, (int*)&(set->fft_size), NULL, 0, 0, 
        NULL, transform_size,1, HIPFFT_R2C, nchan);

  if (status!=HIPFFT_SUCCESS) {
       printf ("Plan failed:");
       if (status==HIPFFT_ALLOC_FAILED) printf("HIPFFT_ALLOC_FAILED");
       if (status==HIPFFT_INVALID_VALUE) printf ("HIPFFT_INVALID_VALUE");
       if (status==HIPFFT_INTERNAL_ERROR) printf ("HIPFFT_INTERNAL_ERROR");
       if (status==HIPFFT_SETUP_FAILED) printf ("HIPFFT_SETUP_FAILED");
       if (status==HIPFFT_INVALID_SIZE) printf ("HIPFFT_INVALID_SIZE");
       printf("\n");
       exit(1);
  }
  printf ("Setting up CUDA streams & events\n");
  gc->nstreams=set->cuda_streams;
  gc->threads=set->cuda_threads;
  if (gc->nstreams<1) {
    printf ("Cannot really work with less than one stream.\n");
    exit(1);
  }
  gc->streams=malloc(gc->nstreams*sizeof(hipStream_t));
  gc->eStart=malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDoneCopy=malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDoneFloatize=malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDoneFFT=malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDonePost=malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDoneCopyBack=malloc(gc->nstreams*sizeof(hipEvent_t));
  hipEvent_t* eStart=(hipEvent_t*)(gc->eStart);
  hipEvent_t* eDoneCopy=(hipEvent_t*)(gc->eDoneCopy);
  hipEvent_t* eDoneFloatize=(hipEvent_t*)(gc->eDoneFloatize);
  hipEvent_t* eDoneFFT=(hipEvent_t*)(gc->eDoneFFT);
  hipEvent_t* eDonePost=(hipEvent_t*)(gc->eDonePost);
  hipEvent_t* eDoneCopyBack=(hipEvent_t*)(gc->eDoneCopyBack);

  for (int i=0;i<gc->nstreams;i++) {
    CHK(hipEventCreate(&eStart[i]));
    CHK(hipEventCreate(&eDoneCopy[i]));
    CHK(hipEventCreate(&eDoneFloatize[i]));
    CHK(hipEventCreate(&eDoneFFT[i]));
    CHK(hipEventCreate(&eDonePost[i]));
    CHK(hipEventCreate(&eDoneCopyBack[i]));
  }
  gc->fstream=gc->bstream=gc->active_streams=0;
  printf ("GPU ready.\n");
}




/**
 * CUDA Kernel byte->float, 1 channel version
 *
 */
__global__ void floatize_1chan(int8_t* sample, hipfftReal* fsample)  {
    int i = FLOATIZE_X*(blockDim.x * blockIdx.x + threadIdx.x);
    for (int j=0; j<FLOATIZE_X; j++) fsample[i+j]=float(sample[i+j]);
}

__global__ void floatize_2chan(int8_t* sample, hipfftReal* fsample1, hipfftReal* fsample2)  {
    int i = FLOATIZE_X*(blockDim.x * blockIdx.x + threadIdx.x);
    for (int j=0; j<FLOATIZE_X/2; j++) {
      fsample1[i/2+j]=float(sample[i+2*j]);
      fsample2[i/2+j]=float(sample[i+2*j+1]);
    }
}



/**
 * CUDA reduction sum
 * we will take bsize complex numbers starting at ffts[istart+bsize*blocknumber]
 * and their copies in NCHUNS, and add the squares
 **/


__global__ void ps_reduce(hipfftComplex *ffts, float* output_ps, size_t istart, size_t avgsize) {
  int tid=threadIdx.x; // thread
  int bl=blockIdx.x; // block, ps bin #
  int nth=blockDim.x; //nthreads
  __shared__ float work[1024];
  //global pos
  size_t pos=istart+bl*avgsize;
  size_t pose=pos+avgsize;
  pos+=tid;
  work[tid]=0;
  while (pos<pose) {
    work[tid]+=ffts[pos].x*ffts[pos].x+ffts[pos].y*ffts[pos].y;
    pos+=nth;
  }
  // now do the tree reduce.
  int csum=nth/2;
  while (csum>0) {
    __syncthreads();
    if (tid<csum) {
      work[tid]+=work[tid+csum];
    }
    csum/=2;
  }
  if (tid==0) output_ps[bl]=work[0];
}
  


/** 
 * CROSS power spectrum reducer
 **/
__global__ void ps_X_reduce(hipfftComplex *fftsA, hipfftComplex *fftsB, 
			    float* output_ps_real, float* output_ps_imag, size_t istart, size_t avgsize) {
  int tid=threadIdx.x; // thread
  int bl=blockIdx.x; // block, ps bin #
  int nth=blockDim.x; //nthreads
  __shared__ float workR[1024];
  __shared__ float workI[1024];
  //global pos
  size_t pos=istart+bl*avgsize;
  size_t pose=pos+avgsize;
  pos+=tid;
  workR[tid]=0;
  workI[tid]=0;
  while (pos<pose) {
    workR[tid]+=fftsA[pos].x*fftsB[pos].x+fftsA[pos].y*fftsB[pos].y;
    workI[tid]+=fftsA[pos].x*fftsB[pos].y-fftsA[pos].y*fftsB[pos].x;
    pos+=nth;
  }
  // now do the tree reduce.
  int csum=nth/2;
  while (csum>0) {
    __syncthreads();
    if (tid<csum) {
      workR[tid]+=workR[tid+csum];
      workI[tid]+=workI[tid+csum];
    }
    csum/=2;
  }
  if (tid==0) {
    output_ps_real[bl]=workR[0];
    output_ps_imag[bl]=workI[0];
  }
}
  



void printDt (hipEvent_t cstart, hipEvent_t cstop) {
  float gpu_time;
  CHK(hipEventElapsedTime(&gpu_time, cstart, cstop));
  printf (" %3.2fms ", gpu_time);
}

void printTiming(GPUCARD *gc, int i) {
  printf ("GPU timing (copy/floatize/fft/post/copyback): ");
  hipEvent_t* eStart=(hipEvent_t*)(gc->eStart);
  hipEvent_t* eDoneCopy=(hipEvent_t*)(gc->eDoneCopy);
  hipEvent_t* eDoneFloatize=(hipEvent_t*)(gc->eDoneFloatize);
  hipEvent_t* eDoneFFT=(hipEvent_t*)(gc->eDoneFFT);
  hipEvent_t* eDonePost=(hipEvent_t*)(gc->eDonePost);
  hipEvent_t* eDoneCopyBack=(hipEvent_t*)(gc->eDoneCopyBack);
  printDt (eStart[i], eDoneCopy[i]);
  printDt (eDoneCopy[i], eDoneFloatize[i]);
  printDt (eDoneFloatize[i], eDoneFFT[i]);
  printDt (eDoneFFT[i], eDonePost[i]);
  printDt (eDonePost[i], eDoneCopyBack[i]);
  tprintfn ("  ");
}

 bool gpuProcessBuffer(GPUCARD *gc, int8_t *buf, WRITER *wr, SETTINGS *set) {
  // pointers and vars
  int8_t** cbuf=(int8_t**)(gc->cbuf);
  hipfftReal** cfbuf=(hipfftReal**)(gc->cfbuf);
  hipfftComplex** cfft=(hipfftComplex**)(gc->cfft);
  float** coutps=(float**)(gc->coutps);

  hipEvent_t* eStart=(hipEvent_t*)(gc->eStart);
  hipEvent_t* eDoneCopy=(hipEvent_t*)(gc->eDoneCopy);
  hipEvent_t* eDoneFloatize=(hipEvent_t*)(gc->eDoneFloatize);
  hipEvent_t* eDoneFFT=(hipEvent_t*)(gc->eDoneFFT);
  hipEvent_t* eDonePost=(hipEvent_t*)(gc->eDonePost);
  hipEvent_t* eDoneCopyBack=(hipEvent_t*)(gc->eDoneCopyBack);
  hipStream_t* streams=(hipStream_t*)gc->streams;

  if (gc->nstreams==1) {
    /// non-streamed version
    hipEventRecord(eStart[0], 0);
    CHK(hipMemcpy(cbuf[0], buf, gc->bufsize , hipMemcpyHostToDevice));
    hipEventRecord(eDoneCopy[0], 0);
    int threadsPerBlock = gc->threads;
    int blocksPerGrid = gc->bufsize / threadsPerBlock/FLOATIZE_X;
    if (gc->nchan==1) 
      floatize_1chan<<<blocksPerGrid, threadsPerBlock >>>(cbuf[0],cfbuf[0]);
    else 
      floatize_2chan<<<blocksPerGrid, threadsPerBlock >>>(cbuf[0],cfbuf[0],&(cfbuf[0][gc->fftsize]));
    CHK(hipGetLastError());

    hipEventRecord(eDoneFloatize[0], 0);
    int status=hipfftExecR2C(gc->plan, cfbuf[0], cfft[0]);
    hipEventRecord(eDoneFFT[0], 0);
    if (status!=HIPFFT_SUCCESS) {
      printf("CUFFT FAILED\n");
      exit(1);
    }    

    if (gc->nchan==1) {
      int psofs=0;
      for (int i=0; i<gc->ncuts; i++) {
	ps_reduce<<<gc->pssize[i], 1024>>> (&cfft[0][0], &(coutps[0][psofs]), gc->ndxofs[i], gc->fftavg[i]);
	psofs+=gc->pssize[i];
      }
    } else {
      // note we need to take into account the tricky N/2+1 FFT size while we do N/2 binning
      // pssize+2 = transformsize+1
      // note that pssize is the full *nchan pssize
      int psofs=0;
      for (int i=0; i<gc->ncuts; i++) {
	ps_reduce<<<gc->pssize1[i], 1024>>> (&cfft[0][0], &(coutps[0][psofs]), gc->ndxofs[i], gc->fftavg[i]);
	psofs+=gc->pssize1[i];
	ps_reduce<<<gc->pssize1[i], 1024>>> (&cfft[0][(gc->fftsize/2+1)], 
                                         &(coutps[0][psofs]), gc->ndxofs[i], gc->fftavg[i]);
	psofs+=gc->pssize1[i];
	ps_X_reduce<<<gc->pssize1[i], 1024>>> (&cfft[0][0], &cfft[0][(gc->fftsize/2+1)], 
					  &(coutps[0][psofs]), &(coutps[0][psofs+gc->pssize1[i]]),
					  gc->ndxofs[i], gc->fftavg[i]);
	psofs+=2*gc->pssize1[i];
      }
    }
    CHK(hipGetLastError());
    hipEventRecord(eDonePost[0], 0);
    CHK(hipMemcpy(gc->outps,coutps[0], gc->tot_pssize*sizeof(float), hipMemcpyDeviceToHost));
    hipEventRecord(eDoneCopyBack[0], 0);
    hipDeviceSynchronize();
    printTiming(gc,0);
    if (set->print_meanvar) {
      // now find some statistic over subsamples of samples
      uint32_t bs=gc->bufsize;
      uint32_t step=gc->bufsize/(32768);
      float fac=bs/step;
      float m1=0.,m2=0.,v1=0.,v2=0.;
      for (int i=0; i<bs; i+=step) {
	float n=buf[i];
	m1+=n; v1+=n*n;
	n=buf[i+1];
	m2+=n; v2+=n*n;
      }
      m1/=fac; v1=sqrt(v1/fac-m1*m1);
      m2/=fac; v2=sqrt(v2/fac-m1*m1);
      tprintfn ("CH1 min/rms: %f %f   CH2 min/rms: %f %f   ",m1,v1,m2,v2);
    }
    if (set->print_maxp) {
      // find max power in each cutout in each channel.
      int of1=0; // CH1 auto
      for (int i=0; i<gc->ncuts; i++) {
	float ch1p=0, ch2p=0;
	int ch1i=0, ch2i=0;
	int of2=of1+gc->pssize1[i]; //CH2 auto
	for (int j=0; j<gc->pssize1[i];j++) {
	  if (gc->outps[of1+j] > ch1p) {ch1p=gc->outps[of1+j]; ch1i=j;}
	  if (gc->outps[of2+j] > ch2p) {ch2p=gc->outps[of2+j]; ch2i=j;}
	}
	of1+=gc->pssize[i];  // next cutout 
	float numin=set->nu_min[i];
	float nustep=(set->nu_max[i]-set->nu_min[i])/(gc->pssize1[i]);
	float ch1f=(numin+nustep*(0.5+ch1i))/1e6;
	float ch2f=(numin+nustep*(0.5+ch2i))/1e6;
	tprintfn ("Peak pow (cutout %i): CH1 %f at %f MHz;   CH2 %f at %f MHz  ",i,log(ch1p),ch1f,log(ch2p),ch2f);
      }
    }
    writerWritePS(wr,gc->outps);
  } else {
    // streamed version
    printf ("Streamed version not ready.\n");
    exit(1);
    
    // first check if there are buffers to store
    while (gc->active_streams>0) {
      // process done streams
      // IMPLEMENT
    }
    // add a new stream
    gc->active_streams++;
    int csi=gc->bstream = (++gc->bstream)%(gc->nstreams);
    hipStream_t cs= streams[gc->bstream];
    hipEventRecord(eStart[csi], cs);
    CHK(hipMemcpyAsync(cbuf[csi], buf, gc->bufsize , hipMemcpyHostToDevice,cs));
    hipEventRecord(eDoneCopy[csi], cs);
    int threadsPerBlock = gc->threads;
    int blocksPerGrid = gc->bufsize / threadsPerBlock/FLOATIZE_X;
    if (gc->nchan==1) 
      floatize_1chan<<<blocksPerGrid, threadsPerBlock, 0, cs>>>(cbuf[csi],cfbuf[csi]);
    else 
      floatize_2chan<<<blocksPerGrid, threadsPerBlock, 0, cs>>>(cbuf[csi],cfbuf[csi],&(cfbuf[csi][gc->fftsize]));
    hipEventRecord(eDoneFloatize[csi], cs);
    int status=hipfftExecR2C(gc->plan, cfbuf[csi], cfft[csi]);
  
    hipEventRecord(eDoneFFT[csi], cs);
    hipEventRecord(eDonePost[csi], cs);
  }



  
  return true;
}
