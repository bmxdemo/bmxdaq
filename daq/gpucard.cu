#include "hip/hip_runtime.h"
/***********************************
***********************************
CUDA PART
***********************************
**********************************/

#define CUDA_COMPILE //to enable cuda types in gpucard.h
#include "gpucard.h"
#undef CUDA_COMPILE
#include "terminal.h"
#include "reduction.h"
#include <memory.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>

#include <iostream>
#include <time.h>
#define FLOATIZE_X 2

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
  if (err != hipSuccess) {
    printf( "CUDA fail: %s in %s at line %d\n", hipGetErrorString( err ),
              file, line );
      exit(1);
  }
}
#define CHK( err ) (HandleError( err, __FILE__, __LINE__ ))

//Print GPU properties
//inputs:
//       prop: pointer to structure containing device properties
//       dev: device number
void printDeviceProperties(hipDeviceProp_t * prop, int dev){
  CHK(hipGetDeviceProperties(prop, dev));
  printf("\nGPU properties \n====================\n");
  printf("Version number:                %d.%d\n",  prop->major, prop->minor);
  printf("Name:                          %s\n",  prop->name);
  printf("Total global memory:           %lu\n",  prop->totalGlobalMem);
  printf("Total shared memory per block: %lu\n",  prop->sharedMemPerBlock);
  printf("Total registers per block:     %d\n",  prop->regsPerBlock);
  printf("Warp size:                     %d\n",  prop->warpSize);
  printf("Maximum memory pitch:          %lu\n",  prop->memPitch);
  printf("Maximum threads per block:     %d\n",  prop->maxThreadsPerBlock);
  for (int i = 0; i < 3; ++i)
  printf("Maximum dimension %d of block:  %d\n", i, prop->maxThreadsDim[i]);
  for (int i = 0; i < 3; ++i)
  printf("Maximum dimension %d of grid:   %d\n", i, prop->maxGridSize[i]);
  printf("Clock rate:                    %d\n",  prop->clockRate);
  printf("Total constant memory:         %lu\n",  prop->totalConstMem);
  printf("Texture alignment:             %lu\n",  prop->textureAlignment);
  printf("Concurrent copy and execution: %s\n",  (prop->deviceOverlap ? "Yes" : "No"));
  printf("Number of multiprocessors:     %d\n",  prop->multiProcessorCount);
  printf("Kernel execution timeout:      %s\n\n",  (prop->kernelExecTimeoutEnabled ? "Yes" : "No"));
}

//Initialize instance of GPUCARD
//Input:
//      gc: instance of GPUCARD to initialize
//      set: settings
void gpuCardInit (GPUCARD *gc, SETTINGS *set) {
  //print out gpu device properties
  gc->devProp = (hipDeviceProp_t *)malloc(sizeof(hipDeviceProp_t));
  printDeviceProperties(gc->devProp, 0);  
  
  int nchan=gc->nchan=1+(set->channel_mask==3);
  if ((nchan==2) and (FLOATIZE_X%2==1)) {
    printf ("Need FLOATIZE_X even for two channels\n");
    exit(1);
  }

  if(!(OPS_PER_THREAD>0) ||  !((OPS_PER_THREAD & (OPS_PER_THREAD-1)) == 0)){
    printf("Need OPS_PER_THREAD to be a power of 2.\n");
    exit(1);
  }

  printf ("\n\nInitializing GPU\n");
  printf ("====================\n");
  printf ("Allocating GPU buffers\n");
  
  int nStreams=set->cuda_streams;
  int nCards=(set->card_mask==3) + 1;
  gc->fftsize=set->fft_size;
  uint32_t bufsize=gc->bufsize=set->fft_size*nchan;
  uint32_t transform_size=gc->transform_size=(set->fft_size/2+1);
  float nunyq=set->sample_rate/2;
  float dnu=nunyq/(set->fft_size/2+1);
  gc->tot_pssize=0;
  gc->ncuts=set->n_cuts;
  for (int i=0; i<gc->ncuts; i++) {
    printf ("Cutout %i:\n",i);
    gc->fftavg[i]=set->fft_avg[i];
    // first sort  reflections etc.
    float numin, numax;
    numin=set->nu_min[i];
    numax=set->nu_max[i];
    while (fabs(numin)>nunyq) numin-=set->sample_rate;
    while (fabs(numax)>nunyq) numax-=set->sample_rate;
    numin=abs(numin);
    numax=abs(numax);
    if (numax<numin) { float t=numin; numin=numax; numax=t; }
    printf ("   Frequencies %f - %f Mhz appear as %f - %f \n",set->nu_min[i]/1e6, set->nu_max[i]/1e6,
      numin/1e6, numax/1e6);
    int imin=int(numin/dnu);
    if (imin==0) imin=1;
    int imax=int(numax/dnu)+1;
    gc->pssize1[i]=(imax-imin)/set->fft_avg[i];
    gc->ndxofs[i]=imin;
    if ((imax-imin)%set->fft_avg[i]>0) gc->pssize1[i]+=1;
    imax=imin+gc->pssize1[i]*set->fft_avg[i];
    numin=imin*dnu;
    numax=imax*dnu;
    set->nu_min[i]=numin;
    set->nu_max[i]=numax;
    set->pssize[i]=gc->pssize1[i];
    if (nchan==2){ 
      if (nCards==1)
	gc->pssize[i]=gc->pssize1[i]*4; // for two channels and two crosses
	else
	gc->pssize[i]=gc->pssize1[i]*16; // for 4 channels and 6*2 crosses
    } else {
      gc->pssize[i]=gc->pssize1[i]*nCards; // just one power spectrum
    }

    gc->tot_pssize+=gc->pssize[i];
    printf ("   Actual freq range: %f - %f MHz (edges!)\n",numin/1e6, numax/1e6);
    printf ("   # PS offset, #PS bins: %i %i\n",gc->ndxofs[i],gc->pssize1[i]);
  }
  CHK(hipHostAlloc(&gc->outps, gc->tot_pssize*sizeof(float), hipHostMallocDefault));

  //allocating GPU buffers
  gc->cbuf=(int8_t***)malloc(nStreams*sizeof(int8_t**));
  gc->cfbuf=(hipfftReal**)malloc(nStreams*sizeof(hipfftReal*));
  gc->cfft=(hipfftComplex**)malloc(nStreams*sizeof(hipfftComplex*));
  gc->coutps=(float**)malloc(nStreams*sizeof(float*));
  for (int i=0;i<nStreams;i++) {
    gc->cbuf[i]=(int8_t**)malloc(nCards*sizeof(int8_t*));
    for(int j=0; j<nCards; j++)
      CHK(hipMalloc(&(gc->cbuf[i][j]),bufsize));
    CHK(hipMalloc(&gc->cfbuf[i], bufsize*nCards*sizeof(hipfftReal)));
    CHK(hipMalloc(&gc->cfft[i],transform_size*nchan*nCards*sizeof(hipfftComplex)));
    CHK(hipMalloc(&gc->coutps[i],gc->tot_pssize*sizeof(float)));
  }

  printf ("Setting up CUFFT\n");
  int status=hipfftPlanMany(&gc->plan, 1, (int*)&(set->fft_size), NULL, 0, 0, 
        NULL, 2*transform_size,1, HIPFFT_R2C, nchan);

  if (status!=HIPFFT_SUCCESS) {
    printf ("Plan failed:");
    if (status==HIPFFT_ALLOC_FAILED) printf("HIPFFT_ALLOC_FAILED");
    if (status==HIPFFT_INVALID_VALUE) printf ("HIPFFT_INVALID_VALUE");
    if (status==HIPFFT_INTERNAL_ERROR) printf ("HIPFFT_INTERNAL_ERROR");
    if (status==HIPFFT_SETUP_FAILED) printf ("HIPFFT_SETUP_FAILED");
    if (status==HIPFFT_INVALID_SIZE) printf ("HIPFFT_INVALID_SIZE");
    printf("\n");
    exit(1);
  }
  printf ("Setting up CUDA streams & events\n");
  gc->nstreams = set->cuda_streams;
  gc->threads=set->cuda_threads;
  if (gc->nstreams<1) {
    printf ("Cannot really work with less than one stream.\n");
    exit(1);
  }
  gc->streams=(hipStream_t*)malloc(gc->nstreams*sizeof(hipStream_t));
  gc->eStart=(hipEvent_t*)malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDoneCopy=(hipEvent_t*)malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDoneFloatize=(hipEvent_t*)malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDoneFFT=(hipEvent_t*)malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDonePost=(hipEvent_t*)malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDoneCopyBack=(hipEvent_t*)malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eBeginCopyBack=(hipEvent_t*)malloc(gc->nstreams*sizeof(hipEvent_t));
  for (int i=0;i<gc->nstreams;i++) {
    //create stream
    CHK(hipStreamCreate(&gc->streams[i]));
    //create events for stream
    CHK(hipEventCreate(&gc->eStart[i]));
    CHK(hipEventCreate(&gc->eDoneCopy[i]));
    CHK(hipEventCreate(&gc->eDoneFloatize[i]));
    CHK(hipEventCreate(&gc->eDoneFFT[i]));
    CHK(hipEventCreate(&gc->eDonePost[i]));
    CHK(hipEventCreate(&gc->eDoneCopyBack[i]));
    CHK(hipEventCreate(&gc->eBeginCopyBack[i]));
  }
 
  gc->fstream = 0; //oldest running stream
  gc->bstream = -1; //newest stream (will become 0 when we actually start with first real stream)
  gc->active_streams = 0; //number of streams currently running
  
  printf ("GPU ready.\n");
}

//Convert bytes to floats, 1 channel version
//Inputs:
//   sample: array of bytes
//       fsample: array of floats to put output in
__global__ void floatize_1chan(int8_t* sample, hipfftReal* fsample)  {
  int i = FLOATIZE_X*(blockDim.x * blockIdx.x + threadIdx.x);
  for (int j=0; j<FLOATIZE_X; j++) fsample[i+j]=float(sample[i+j]);
}


//Convert bytes to floats, 2 channel version
//Inputs:
//   sample: array of bytes with the 2 channels interleaved
//       fsample1: array of floats to put converted bytes from channel 1 in
//       fsample2: array of floats to put converted bytes from channel 2 in
__global__ void floatize_2chan(int8_t* sample, hipfftReal* fsample1, hipfftReal* fsample2)  {
  int i = FLOATIZE_X*(blockDim.x * blockIdx.x + threadIdx.x);
  for (int j=0; j<FLOATIZE_X/2; j++) {
    fsample1[i/2+j]=float(sample[i+2*j]);
    fsample2[i/2+j]=float(sample[i+2*j+1]);
  }
}

//Print the elapsed time between 2 cuda events
void printDt (hipEvent_t cstart, hipEvent_t cstop, float * total, TWRITER * t) {
  float gpu_time;
  CHK(hipEventElapsedTime(&gpu_time, cstart, cstop));
  tprintfn (t, 0, " %3.2fms ", gpu_time);
  *total +=gpu_time;
}

void printTiming(GPUCARD *gc, int i, TWRITER * t) {
  float totalTime = 0;
  tprintfn (t, 0, "GPU timing (copy/floatize/fft/post/copyback): ");
  printDt (gc->eStart[i], gc->eDoneCopy[i], &totalTime, t);
  printDt (gc->eDoneCopy[i], gc->eDoneFloatize[i], &totalTime, t);
  printDt (gc->eDoneFloatize[i], gc->eDoneFFT[i], &totalTime, t);
  printDt (gc->eDoneFFT[i], gc->eDonePost[i], &totalTime, t);
  printDt (gc->eBeginCopyBack[i], gc->eDoneCopyBack[i], &totalTime, t);
  tprintfn (t,1,"");
  tprintfn (t, 1, "GPU timing total: %3.2f ", totalTime);
}


void printLiveStat(SETTINGS *set, GPUCARD *gc, int8_t **buf, TWRITER *twr) {
  int nCards=(set->card_mask==3) + 1;

  if (set->print_meanvar) {
    // now find some statistic over subsamples of samples
    uint32_t bs=gc->bufsize;
    uint32_t step=gc->bufsize/(32768);
    float NSub=bs/step; // number of subsamples to take
    float m1=0.,m2=0.,v1=0.,v2=0.;
    float m3=0.,m4=0.,v3=0.,v4=0.;
    for (int i=0; i<bs; i+=step) { // take them in steps of step
      float n=buf[0][i];
      m1+=n; v1+=n*n;
      n=buf[0][i+1];
      m2+=n; v2+=n*n;
      if (nCards==2) {
	n=buf[1][i];        
	m3+=n; v3+=n*n;
	n=buf[1][i+1];
	m4+=n; v4+=n*n;

      }
    }
    m1/=NSub; v1=sqrt(v1/NSub-m1*m1); //mean and variance
    m2/=NSub; v2=sqrt(v2/NSub-m2*m2);
    tprintfn (twr,1,"CH1 mean/rms: %f %f   CH2 mean/rms: %f %f   ",m1,v1,m2,v2);
    if (nCards==2) {
      m3/=NSub; v3=sqrt(v3/NSub-m3*m3); //mean and variance
      m4/=NSub; v4=sqrt(v4/NSub-m4*m4);
      tprintfn (twr,1,"CH3 mean/rms: %f %f   CH4 mean/rms: %f %f   ",m3,v3,m4,v4);
    }
  }
  if (set->print_maxp) {
    // find max power in each cutout in each channel.
    int of1=0; // CH1 auto

    for (int i=0; i<gc->ncuts; i++) {
      int of2=of1+gc->pssize1[i]; //CH2 auto 
      int of3=of1+2*gc->pssize1[i]; // CH3 auto
      int of4=of1+3*gc->pssize1[i]; // CH4 auto

      float ch1p=0, ch2p=0, ch3p=0, ch4p=0;
      int ch1i=0, ch2i=0, ch3i=0, ch4i=0;

      for (int j=0; j<gc->pssize1[i];j++) {
	if (gc->outps[of1+j] > ch1p) {ch1p=gc->outps[of1+j]; ch1i=j;}
	if (gc->outps[of2+j] > ch2p) {ch2p=gc->outps[of2+j]; ch2i=j;}
	if (nCards==2) {
	  if (gc->outps[of3+j] > ch3p) {ch3p=gc->outps[of3+j]; ch3i=j;}
	  if (gc->outps[of4+j] > ch4p) {ch4p=gc->outps[of4+j]; ch4i=j;}
	}
      }
      of1+=gc->pssize[i];  // next cutout 
      float numin=set->nu_min[i];
      float nustep=(set->nu_max[i]-set->nu_min[i])/(gc->pssize1[i]);
      float ch1f=(numin+nustep*(0.5+ch1i))/1e6;
      float ch2f=(numin+nustep*(0.5+ch2i))/1e6;
      tprintfn (twr,1,"Peak pow (cutout %i): CH1 %f at %f MHz;   CH2 %f at %f MHz  ",
		i,log(ch1p),ch1f,log(ch2p),ch2f);
      if (nCards==2) {
	float ch3f=(numin+nustep*(0.5+ch3i))/1e6;
	float ch4f=(numin+nustep*(0.5+ch4i))/1e6;
	tprintfn (twr,1,"Peak pow (cutout %i): CH3 %f at %f MHz;   CH4 %f at %f MHz  ",
		  i,log(ch3p),ch3f,log(ch4p),ch4f);
      }
    }
  }

}

//Process one data packet from the digitizer
//Input:
//  gc: graphics card
//      buf: data from digitizer
//      wr: writer to write out power spectra and outliers to files

//  set: settings
int gpuProcessBuffer(GPUCARD *gc, int8_t **buf, WRITER *wr, TWRITER *twr, SETTINGS *set) {
  //streamed version
  //Check if other streams are finished and proccess the finished ones in order (i.e. print output to file)

  int nCards=(set->card_mask==3) + 1;

  while(gc->active_streams > 0){
    // printf ("S:%i ", hipEventQuery(gc->eStart[gc->fstream])==hipSuccess);
    // printf ("%i ", hipEventQuery(gc->eDoneCopy[gc->fstream])==hipSuccess);
    // printf ("%i ", hipEventQuery(gc->eDoneFloatize[gc->fstream])==hipSuccess);
    // printf ("%i ", hipEventQuery(gc->eDoneFFT[gc->fstream])==hipSuccess);
    // printf ("%i [%i]\n ", hipEventQuery(gc->eDonePost[gc->fstream])==hipSuccess, gc->fstream);
    if(hipEventQuery(gc->eDonePost[gc->fstream])==hipSuccess){

      //print time and write to file
      hipEventRecord(gc->eBeginCopyBack[gc->fstream], gc->streams[gc->fstream]);
      CHK(hipMemcpyAsync(gc->outps,gc->coutps[gc->fstream], 
          gc->tot_pssize*sizeof(float), hipMemcpyDeviceToHost, gc->streams[gc->fstream]));
      hipEventRecord(gc->eDoneCopyBack[gc->fstream], gc->streams[gc->fstream]);
      //hipDeviceSynchronize();
      hipEventSynchronize(gc->eDoneCopyBack[gc->fstream]);
      printTiming(gc,gc->fstream,twr);
      printLiveStat(set,gc,buf,twr);
      writerAccumulatePS(wr,gc->outps,twr);
      gc->fstream = (++gc->fstream)%(gc->nstreams);
      gc->active_streams--;
    }
    else 
      break;      
  }  

  if(gc->active_streams == gc->nstreams){ //if no empty streams
       	return false;
  }
  gc->active_streams++;
  int csi = gc->bstream = (++gc->bstream)%(gc->nstreams); //add new stream

  hipStream_t cs= gc->streams[gc->bstream];
  hipEventRecord(gc->eStart[csi], cs);
  

  //memory copy
  for(int i=0; i<nCards; i++)
    CHK(hipMemcpyAsync(gc->cbuf[csi][i], buf[i], gc->bufsize , hipMemcpyHostToDevice,cs));
    
  //floatize
  hipEventRecord(gc->eDoneCopy[csi], cs);
  int threadsPerBlock = gc->threads;
  int blocksPerGrid = gc->bufsize / threadsPerBlock/FLOATIZE_X;
  if (gc->nchan==1) 
    floatize_1chan<<<blocksPerGrid, threadsPerBlock, 0, cs>>>(gc->cbuf[csi][0],gc->cfbuf[csi]);
  else
    for(int i=0; i<nCards; i++)
      floatize_2chan<<<blocksPerGrid, threadsPerBlock, 0, cs>>>
        (gc->cbuf[csi][i],&(gc->cfbuf[csi][gc->fftsize*2*i]),&(gc->cfbuf[csi][gc->fftsize*(2*i+1)]));
  hipEventRecord(gc->eDoneFloatize[csi], cs);
  
  //perform fft
  int status = hipfftSetStream(gc->plan, cs);
  if(status !=HIPFFT_SUCCESS) {
    printf("CUFFTSETSTREAM failed\n");
    exit(1);
  }
  for(int i=0; i<nCards;i++){
    status=hipfftExecR2C(gc->plan, &(gc->cfbuf[csi][gc->bufsize*i]), &(gc->cfft[csi][2*i*gc->transform_size]));
    if (status!=HIPFFT_SUCCESS) {
      printf("CUFFT FAILED\n");
      exit(1);
    } 
  } 
  hipEventRecord(gc->eDoneFFT[csi], cs);
  
  //compute spectra
  if (gc->nchan==1) {
    int psofs=0;
    for (int i=0; i<gc->ncuts; i++) {
      ps_reduce<<<gc->pssize[i], 1024, 0, cs>>> (gc->cfft[csi], &(gc->coutps[csi][psofs]), 
          gc->ndxofs[i], gc->fftavg[i]);
      psofs+=gc->pssize[i];
    }
  } 
  else if(gc->nchan==2){
    // note we need to take into account the tricky N/2+1 FFT size while we do N/2 binning
    // pssize+2 = transformsize+1
        
    int psofs=0;
    for (int i=0; i<gc->ncuts; i++) {
   
      for(int j=0; j<nCards; j++){
        ps_reduce<<<gc->pssize1[i], 1024, 0, cs>>> (&gc->cfft[csi][2*j*gc->transform_size], 
            &(gc->coutps[csi][psofs]), gc->ndxofs[i], gc->fftavg[i]);
        psofs+=gc->pssize1[i];
        
        ps_reduce<<<gc->pssize1[i], 1024, 0, cs>>> (&gc->cfft[csi][(2*j+1)*gc->transform_size], 
            &(gc->coutps[csi][psofs]), gc->ndxofs[i], gc->fftavg[i]);
        psofs+=gc->pssize1[i];
      }
      //cross spectra
      for(int j = 0; j<nCards*2; j++)
        for(int k = j+1; k < nCards*2 ; k++){
         //NEED TO CHECK THAT PARAMETERS ARE ALL CORRECT FOR TWO CARDS AND FOR ONE CARD....
	  ps_X_reduce<<<gc->pssize1[i], 1024, 0, cs>>> (&gc->cfft[csi][j*gc->transform_size], 
							&gc->cfft[csi][k*gc->transform_size], 
	    &(gc->coutps[csi][psofs]), &(gc->coutps[csi][psofs+gc->pssize1[i]]),
            gc->ndxofs[i], gc->fftavg[i]);
          psofs+=2*gc->pssize1[i];
        }
    }
  }
  else{
    printf("Can only handle 1 or 2 channels\n");
    exit(1);
  }
 
  CHK(hipGetLastError());
  hipEventRecord(gc->eDonePost[csi], cs);
      
  return true;
}
